#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "header.h"

// ======================== macros for allocating device memory and error check

#define CUDA_ERR_CHECK(err,msg) (\
		{if (err != hipSuccess) { \
			fprintf(stderr, msg " - %s\n", hipGetErrorString(err)); \
			exit(EXIT_FAILURE); \
		} \
	})

#define CUDA_MEM_INIT(dest, src, size, type) {\
	hipError_t err = hipSuccess;\
	size_t  arrSize = size * sizeof(type);\
	err = hipMalloc((void**)&dest, arrSize);\
	CUDA_ERR_CHECK(err, "Failed to allocate device memory");\
	err = hipMemcpy(dest, src, arrSize, hipMemcpyHostToDevice);\
	CUDA_ERR_CHECK(err, "Failed to copy data from host to device"); }\

	
// ======================== groups
void getNK_CPU(int mutant_num, int seq2_len, int* n, int* k);
void getNK_CPU(int mutant_num, int seq2_len, int* n, int* k)
{
	int i;
	int num_of_mutants_in_row = seq2_len;

	for(i = 1; i < seq2_len; i++){
		if(mutant_num - (num_of_mutants_in_row - 1) > 0){
		    mutant_num -= (num_of_mutants_in_row - 1);
		    num_of_mutants_in_row--;
		}else{
		    break;
		}
	}
	
	*n = i;	
	*k = i + mutant_num;
}

__device__ const char* conservatives[CONSERVATIVES_LEN] = {
	"NDEQ", "NEQK", "STA", "MILV", "QHRK", "NHQK", "FYW", "HY", "MILF"
};

__device__ const char* semi_conservatives[SEMI_CONSERVATIVES_LEN] = {
	"SAG", "ATV", "CSA", "SGND", "STPA", "STNK", "NEQHRK", "NDEQHK", "SNDEQK", "HFY", "FVLIM"
};

// ======================== // check if char in group (same as strchr)

__device__ int isCharExist(const char* s, int c)
{
	do {
	    if (*s == c)
		return 1;
	} while (*s++);
	  
  	return 0;
}

// ========================

__device__ int checkInGroup(char first, char second, const char** group, int len) // checks if both chars in the group
{
	for (int i = 0; i < len; i++)
	{
		if (isCharExist(group[i], first) != NULL
			&& isCharExist(group[i], second) != NULL)
			return 1;
	}
	return 0;
}

// ========================

__device__ float compareChars(char first, char second, float weights[]) // assign suitable weight for chars 
{
	if (first == second)
		return weights[0];
	else if (checkInGroup(first, second, conservatives, CONSERVATIVES_LEN))
		return -weights[1];
	else if (checkInGroup(first, second, semi_conservatives, SEMI_CONSERVATIVES_LEN))
		return -weights[2];
		
	return -weights[3];
}

// ========================

__global__ void calcSimilarityKernel(float* d_similarity, char* d_seq1, char* d_mutant, float* d_weights, int mutant_size) // traverses char by char and checks similarity
{
	//int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	//if(i < mutant_size)
		//d_similarity[i] = compareChars(d_seq1[i], d_mutant[i], d_weights);
}

// ========================

void calcMutantSimilarityCUDA(float* similarity, char* seq1, char* mutant, float weights[]) // entry point to CUDA , allocates memory and calls kernel func
{
	int mutant_size = strlen(mutant);
	
	char* d_seq1 = NULL; // allocate seq1 memory
	CUDA_MEM_INIT(d_seq1, seq1, strlen(seq1), char);
	
	char* d_mutant = NULL; // allocate mutant memory
	CUDA_MEM_INIT(d_mutant, mutant, mutant_size, char);
	
	float* d_weights = NULL; // allocate weights memory
	CUDA_MEM_INIT(d_weights, weights, WEIGHTS_NUM, float);
	
	float* d_similarity = NULL; // allocate similarity memory
	hipMalloc((void**)&d_similarity, mutant_size * sizeof(float));
	
	int threads = (int)ceil(mutant_size);
	if (threads % 32 != 0) // assert threads is multiple of 32
		threads = threads+32 - threads%32; 
	
	int blocks = (mutant_size + threads - 1) / threads;
	
	calcSimilarityKernel<<<blocks, threads>>>(d_similarity, d_seq1, d_mutant, d_weights, mutant_size);

	hipMemcpy(similarity, d_similarity, mutant_size * sizeof(float), hipMemcpyDeviceToHost); // copy result to host memory
	
	hipFree(d_seq1);
	hipFree(d_mutant);
	hipFree(d_weights);
	hipFree(d_similarity);
}

// ========================

__device__ void getNK(int mutant_num, int seq2_len, int* n, int* k)
{
	int i;
	int num_of_mutants_in_row = seq2_len;

	for(i = 1; i < seq2_len; i++){
		if(mutant_num - (num_of_mutants_in_row - 1) > 0){
		    mutant_num -= (num_of_mutants_in_row - 1);
		    num_of_mutants_in_row--;
		}else{
		    break;
		}
	}
	
	*n = i;	
	*k = i + mutant_num;
}

__device__ float calcMutantScore(char* seq1, char* seq2, float* weights, float* d_conservative_matrix,int len2, int n, int k, int index, int offset)
{
	float score = 0;
	int i = 0, j = i;
	for (i = 0; i < len2 - 2; i++, j++)
	{
		if (j == n || j == k) 
			j++;
		float tmp_score = d_conservative_matrix[(seq1[i] - 'A') * 26 + (seq2[j] - 'A')];
		score += tmp_score;
	}	

	return score;	
}

__global__ void calcMutantBestScoreKernel(char* d_seq1, char* d_seq2, float* d_weights, float* d_mutantsBestScores, int* d_mutantsBestOffsets, int num_mutants, int maxOffset, int len2, float* d_conservative_matrix)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int n,k;
	float bestScore = -10000;
	int offset = 0;
	getNK(i+1, len2, &n, &k);
	if (i < num_mutants)
	{
		for (int j = 0; j < maxOffset; j++)
		{
			float score = calcMutantScore(&d_seq1[j], d_seq2, d_weights, d_conservative_matrix, len2, n, k, i, j);
			if (score > bestScore)
			{
				bestScore = score;
				offset = j;	
			}
		}
		d_mutantsBestScores[i]	= bestScore;
		d_mutantsBestOffsets[i] = offset;
	}
}

__global__ void reduction_cuda(float* d_mutantsBestScores, int* d_mutantsBestOffsets, float* d_reductionBestScores, int* d_reductionBestOffsets, int* d_reductionMutantNum, int num_of_elements){
	__shared__ int shared_mutant_num[256];
	__shared__ float shared_mutant_score[256];
	__shared__ float shared_mutant_best_offset[256];
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
	
	// load shared mem from global  mem
	shared_mutant_score[tid] = d_mutantsBestScores[i];
	shared_mutant_num[tid] = i;
	if(i >= num_of_elements)
		shared_mutant_score[tid] = 0;
	__syncthreads();	// make sure entine block is loaded!
	// do reduction in shared mem
	for(int s = blockDim.x / 2; s > 0; s >>= 1){
		if(tid < s){
			if(shared_mutant_score[tid] < shared_mutant_score[tid + s]){
				shared_mutant_score[tid] = shared_mutant_score[tid + s];
				shared_mutant_num[tid] = shared_mutant_num[tid + s];
				shared_mutant_best_offset[tid] = shared_mutant_best_offset[tid + s];
			}
		}
		__syncthreads();	// make sure all adds at ine stage are done!
	}
	// only thread 0 writes result for this block back to global mem
	if(tid == 0){
		d_reductionBestScores[blockIdx.x] = shared_mutant_score[tid];
		d_reductionBestOffsets[blockIdx.x] = shared_mutant_best_offset[tid];
		d_reductionMutantNum[blockIdx.x] = shared_mutant_num[tid];
	}
}

void calcBestScoreCUDA(char* seq1, char* seq2, float* conservative_matrix, float* weights)
{
	hipError_t err = hipSuccess;
	
	int len1 = strlen(seq1);
	int len2 = strlen(seq2);
	int maxOffset = len1 - (len2-2) + 1;
	int num_mutants = len2 * (len2 - 1) / 2;
	
	float* mutantsBestScores = (float*) malloc(num_mutants * sizeof(float));	
	int* mutantsBestOffsets = (int*) malloc(num_mutants * sizeof(int));		
	
	char* d_seq1 = NULL; // allocate seq1 memory
	CUDA_MEM_INIT(d_seq1, seq1, len1, char);
	
	char* d_seq2 = NULL; // allocate seq2 memory
	CUDA_MEM_INIT(d_seq2, seq2, len2, char);
	
	float* d_weights = NULL; // allocate weights memory
	CUDA_MEM_INIT(d_weights, weights, WEIGHTS_NUM, float);
	
	float* d_conservative_matrix = NULL; // allocate weights memory
	CUDA_MEM_INIT(d_conservative_matrix, conservative_matrix, 26 * 26, float);
	
	float* d_mutantsBestScores = NULL; 
	err = hipMalloc((void**)&d_mutantsBestScores, num_mutants*sizeof(float));
	CUDA_ERR_CHECK(err, "Failed to allocate device memory");
	
	int* d_mutantsBestOffsets = NULL;
	err = hipMalloc((void**)&d_mutantsBestOffsets, num_mutants*sizeof(int));
	CUDA_ERR_CHECK(err, "Failed to allocate device memory");
	
	int threads = 256;
	int blocks = (num_mutants + threads-1) / threads;
	calcMutantBestScoreKernel<<<blocks, threads>>>(d_seq1, d_seq2, d_weights, d_mutantsBestScores, d_mutantsBestOffsets, num_mutants, maxOffset, len2, d_conservative_matrix);
	
	hipMemcpy(mutantsBestScores, d_mutantsBestScores, num_mutants * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(mutantsBestOffsets, d_mutantsBestOffsets, num_mutants * sizeof(int), hipMemcpyDeviceToHost);
	
	/*
	float maxScore = -10000;
	int bestOffset = 0;
	int bestMutantNum = -1;
	
	for (int i = 0; i < num_mutants; i++)
	{
		if (mutantsBestScores[i] > maxScore)
		{
			maxScore = mutantsBestScores[i];
			bestOffset = mutantsBestOffsets[i];
			bestMutantNum = i;
		}
	}
	int n,k;
	getNK_CPU(bestMutantNum + 1, len2, &n, &k);
	printf("mutant num: %d, MS(%d,%d), score: %1.2f, offset: %d\n", bestMutantNum, n, k, maxScore, bestOffset);
	*/
	calcBestScoreOmp(mutantsBestScores, mutantsBestOffsets, num_mutants);
	
	free(mutantsBestScores);
	free(mutantsBestOffsets);
	hipFree(d_seq1);
	hipFree(d_seq2);
	hipFree(d_mutantsBestScores);
	hipFree(d_mutantsBestOffsets);
	hipFree(d_conservative_matrix);
	
}
